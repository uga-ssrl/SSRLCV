#include "hip/hip_runtime.h"
#include "SIFT_FeatureFactory.cuh"

ssrlcv::SIFT_FeatureFactory::SIFT_FeatureFactory(float orientationContribWidth, float descriptorContribWidth){
  this->orientationContribWidth = orientationContribWidth;
  this->descriptorContribWidth = descriptorContribWidth;
}

/**
 * @brief This function generates features points from an image.
 * 
 * @param image a pointer to the image object
 * @param dense specifies whether or not dense SIFT is used
 * @param maxOrientations max orientations of the image
 * @param orientationThreshold orientation threshold
 * @return a vector of SIFT feature descriptors
 */
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>> ssrlcv::SIFT_FeatureFactory::generateFeatures(ssrlcv::ptr::value<ssrlcv::Image> image, bool dense, unsigned int maxOrientations, float orientationThreshold){
  logger.info.printf("Generating SIFT features for image %d", image->id);
  ssrlcv::ptr::value<Unity<Feature<SIFT_Descriptor>>> features;
  
  // make sure we are operating in GPU memory
  MemoryState origin = image->pixels->getMemoryState();
  if(origin != gpu) image->pixels->setMemoryState(gpu);

  // convert image to BW
  if(image->colorDepth != 1){
    convertToBW(image->pixels,image->colorDepth);
    image->colorDepth = 1;
  }

  // this conditional is skipped becuase we are NOT using dense SIFT
  if(dense){

    clock_t timer = clock();
    ssrlcv::ptr::value<ssrlcv::Unity<float>> pixelsFLT = convertImageToFlt(image->pixels);
    if(origin != gpu) image->pixels->setMemoryState(origin);//no longer need to force pixels on gpu
    normalizeImage(pixelsFLT);
    ssrlcv::ptr::value<ssrlcv::Unity<float2>> gradients = generatePixelGradients(image->size, pixelsFLT);
    //12x12 border
    ssrlcv::ptr::value<ssrlcv::Unity<float2>> keyPoints = ssrlcv::ptr::value<ssrlcv::Unity<float2>>(nullptr,(image->size.x-24)*(image->size.y-24),cpu);
    for(int y = 0; y < image->size.y-24; ++y){
      for(int x = 0; x < image->size.x-24; ++x){
        keyPoints->host.get()[y*(image->size.x-24) + x] = {(float)x + 12.0f, (float)y + 12.0f};
      }
    }

    // will free CPU memory and instantiate GPU memory
    keyPoints->setMemoryState(gpu);

    logger.info.printf("Dense SIFT prep done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

    features = this->createFeatures(image->size,orientationThreshold,maxOrientations,1.0f,gradients,keyPoints);
  }
  else{
    uint2 scaleSpaceDim = {4,6};
    //int nspo = scaleSpaceDim.y - 3;//num scale space/octave in dog made from a {4,6} ScaleSpace
    float noiseThreshold = 0.01f;//*(powf(2,1.0f/nspo)-1)/(powf(2,1.0f/3.0f)-1);//if 0.15 there is a segfault
    float edgeThreshold = 12.1f;//12.1 = (10.0f + 1)^2 / 10.0f //formula = (r+1)^2/r from lowes paper where r = 10
    
    // instantiate a pointer to a DOG object with the image
    int2 kernelSize = {8,8};
    float2 sigmaMultiplier = {2,sqrtf(2.0f)};
    ssrlcv::ptr::value<DOG> dog = ssrlcv::ptr::value<DOG>(image,-1,scaleSpaceDim,sqrtf(2.0f)/2.0f,sigmaMultiplier,kernelSize,true);//last true specifies dog conversion
    logger.info<<"\tdog created";

    // set memory back to CPU
    if(origin != gpu) image->pixels->setMemoryState(origin);//no longer need to force pixels on gpu
    // std::string dump = "out/dog";
    // dog->dumpData(dump);
    dog->findKeyPoints(noiseThreshold,edgeThreshold,true); 

    ssrlcv::ptr::value<ScaleSpace::Octave> currentOctave(nullptr);
    ssrlcv::ptr::value<ScaleSpace::Octave::Blur> currentBlur;
    int numFeaturesProduced = 0;
    ssrlcv::ptr::host<MemoryState> extremaOrigin(dog->depth.x);
    unsigned int numKeyPointsInBlur = 0;
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};

    for(int o = 0; o < dog->depth.x; ++o){
      currentOctave = dog->octaves.get()[o];
      if(currentOctave->extrema == nullptr) continue;
      extremaOrigin.get()[o] = currentOctave->extrema->getMemoryState(); 
      if(extremaOrigin.get()[o] != gpu) currentOctave->extrema->setMemoryState(gpu);
      
      for(int b = 0; b < dog->depth.y; ++b){
        if(b + 1 == dog->depth.y){
          numKeyPointsInBlur = currentOctave->extrema->size() - currentOctave->extremaBlurIndices.get()[b];
        }
        else{
          numKeyPointsInBlur = currentOctave->extremaBlurIndices.get()[b+1] - currentOctave->extremaBlurIndices.get()[b];
        }
        if(numKeyPointsInBlur == 0) continue;

        currentBlur = currentOctave->blurs.get()[b];
        grid = {1,1,1};
        block = {1,1,1};
        getFlatGridBlock(numKeyPointsInBlur,grid,block,checkKeyPoints);

        checkKeyPoints<<<grid,block>>>(numKeyPointsInBlur,currentOctave->extremaBlurIndices.get()[b],currentBlur->size, currentOctave->pixelWidth,
          this->descriptorContribWidth,currentOctave->extrema->device.get());
        hipDeviceSynchronize();
        CudaCheckError();
      }
      currentOctave->discardExtrema();
    }
    // have not transfered any extrema back to extremaOrigin

    dog->computeKeyPointOrientations(orientationThreshold,maxOrientations,this->orientationContribWidth,true);

    // then create features from each of the keyPoints
    unsigned int numKeyPoints = 0;
    for(int o = 0; o < dog->depth.x; ++o){
      if(dog->octaves.get()[o]->extrema == nullptr) continue;
      numKeyPoints += dog->octaves.get()[o]->extrema->size();
    }
    if(numKeyPoints == 0){
      logger.err<<"ERROR: something went wrong and there are 0 keypoints\n";
      exit(0);
    }
    logger.info.printf("total keypoints found = %d", numKeyPoints);
    logger.info<<"creating features from keypoints...";
    // here, a feature vector containing SIFT feature descriptors is generated for each key point
    features = ssrlcv::ptr::value<Unity<Feature<SIFT_Descriptor>>>(nullptr,numKeyPoints,gpu);
    // fill descriptors based on SSKeyPoint information
    block = {4,4,8};

    MemoryState gradientsOrigin;

    for(int o = 0; o < dog->depth.x; ++o){
      currentOctave = dog->octaves.get()[o];
      if(currentOctave->extrema == nullptr) continue;
      //extrema should already be on gpu from last loop
      for(int b = 0; b < dog->depth.y; ++b){
        if(b + 1 == dog->depth.y){
          numKeyPointsInBlur = currentOctave->extrema->size() - currentOctave->extremaBlurIndices.get()[b];
        }
        else{
          numKeyPointsInBlur = currentOctave->extremaBlurIndices.get()[b+1] - currentOctave->extremaBlurIndices.get()[b];
        }
        if(numKeyPointsInBlur == 0) continue;
  
        currentBlur = currentOctave->blurs.get()[b];
        gradientsOrigin = currentBlur->gradients->getMemoryState();
        
        // set memory state to GPU
        if(gradientsOrigin != gpu) currentBlur->gradients->setMemoryState(gpu);
        // initilize grid var
        grid = {1,1,1};
        // get grid dimensions
        getGrid(numKeyPointsInBlur,grid);
        
        // execute the fillDescriptors kernel, which fills the descriptors
        fillDescriptors<<<grid,block>>>(numKeyPointsInBlur,currentBlur->size, 
          features->device.get() + numFeaturesProduced, currentOctave->pixelWidth, this->descriptorContribWidth,
          currentOctave->extrema->device.get() + currentOctave->extremaBlurIndices.get()[b], currentBlur->gradients->device.get());
        hipDeviceSynchronize();
        CudaCheckError();

        numFeaturesProduced += numKeyPointsInBlur;
        if(gradientsOrigin != gpu) currentBlur->gradients->setMemoryState(gradientsOrigin);
      } // for
      if(extremaOrigin.get()[o] != gpu) currentOctave->extrema->setMemoryState(extremaOrigin.get()[o]);
      logger.info.printf("\tfeatures created from octave %d", o);
    } // for
  }
  return features; 
} // generateFeatures

ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>> ssrlcv::SIFT_FeatureFactory::createFeatures(uint2 imageSize,float orientationThreshold, unsigned int maxOrientations, float pixelWidth, ssrlcv::ptr::value<ssrlcv::Unity<float2>> gradients, ssrlcv::ptr::value<ssrlcv::Unity<float2>> keyPoints){

  clock_t timer = clock();

  ssrlcv::ptr::device<int> thetaNumbers_device(keyPoints->size()*maxOrientations);
  ssrlcv::ptr::device<float> thetas_device(keyPoints->size()*maxOrientations);
  ssrlcv::ptr::host<float> thetas_host(keyPoints->size()*maxOrientations);
  for(int i = 0; i < keyPoints->size()*maxOrientations; ++i){
    thetas_host.get()[i] = -1.0f;
  }
  CudaSafeCall(hipMemcpy(thetas_device.get(),thetas_host.get(),keyPoints->size()*maxOrientations*sizeof(float),hipMemcpyHostToDevice));

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};

  void (*fp)(const unsigned long, const unsigned int,
    const float, const float, const float, const float2*,
    const float2*, int*, const unsigned int, const float,
    float*) = &computeThetas;

  getFlatGridBlock(keyPoints->size(),grid,block,fp);

  computeThetas<<<grid,block>>>(keyPoints->size(),imageSize.x,pixelWidth,
    this->orientationContribWidth,ceil(3.0f*this->orientationContribWidth/pixelWidth),
    keyPoints->device.get(), gradients->device.get(), thetaNumbers_device.get(), maxOrientations,
    orientationThreshold,thetas_device.get());
  hipDeviceSynchronize();
  CudaCheckError();

  logger.info.printf("compute thetas done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);
  timer = clock();

  thrust::device_ptr<int> tN(thetaNumbers_device.get());
  thrust::device_ptr<int> end = thrust::remove(tN, tN + keyPoints->size()*maxOrientations, -1);
  int numFeatures = end - tN;

  thrust::device_ptr<float> t(thetas_device.get());
  thrust::device_ptr<float> new_end = thrust::remove(t, t + keyPoints->size()*maxOrientations, -FLT_MAX);

  logger.info.printf("theta compaction done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);
  timer = clock();

  grid = {1,1,1};
  block = {4,4,8};
  getGrid(numFeatures,grid);

  ssrlcv::ptr::device<Feature<SIFT_Descriptor>> features_device(numFeatures);
  fillDescriptors<<<grid,block>>>(numFeatures,imageSize.x,features_device.get(),pixelWidth,
    this->descriptorContribWidth,ceil(this->descriptorContribWidth/pixelWidth),
    thetas_device.get(),thetaNumbers_device.get(),keyPoints->device.get(),gradients->device.get());
  hipDeviceSynchronize();
  CudaCheckError();

  logger.info.printf("fill descriptors done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  return ssrlcv::ptr::value<Unity<Feature<SIFT_Descriptor>>>(features_device,numFeatures,gpu);
}



/*
CUDA implementations
*/

/*
DEVICE METHODS
*/

//reimplemented as these are inline functions

__device__ __forceinline__ float ssrlcv::getMagnitude(const int2 &vector){
  return sqrtf((float)dotProduct(vector, vector));
}
__device__ __forceinline__ float ssrlcv::getMagnitude(const float2 &vector){
  return sqrtf(dotProduct(vector, vector));
}
__device__ __forceinline__ float ssrlcv::atomicMinFloat (float * addr, float value){
  float old;
  old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float ssrlcv::atomicMaxFloat (float * addr, float value){
  float old;
  old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float ssrlcv::edgeness(const float (&hessian)[2][2]){
    float e = trace(hessian);
    return e*e/determinant(hessian);    
}


/*
KERNELS
*/

__global__ void ssrlcv::computeThetas(const unsigned long numKeyPoints, const unsigned int imageWidth,
    const float pixelWidth, const float lambda, const float windowWidth, const float2* __restrict__ keyPointLocations,
    const float2* gradients, int* __restrict__ thetaNumbers, const unsigned int maxOrientations, const float orientationThreshold,
    float* __restrict__ thetas){

  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;

  if(globalID < numKeyPoints){
    float2 keyPoint = keyPointLocations[globalID];
    float orientationHist[36] = {0.0f};
    float maxHist = 0.0f;
    int regNumOrient = maxOrientations;
    float2 gradient = {0.0f,0.0f};
    float2 temp2 = {0.0f,0.0f};
    float weight = 2*(windowWidth*pixelWidth/3.0f)*(windowWidth*pixelWidth/3.0f);
    float angle = 0.0f;
    float rad10 = pi/18.0f;
    for(float y = (keyPoint.y - windowWidth)/pixelWidth; y <= (keyPoint.y + windowWidth)/pixelWidth; y+=1.0f){
      for(float x = (keyPoint.x - windowWidth)/pixelWidth; x <= (keyPoint.x + windowWidth)/pixelWidth; x+=1.0f){
        gradient = gradients[llroundf(y)*imageWidth + llroundf(x)];//interpolation?
        temp2 = {x*pixelWidth - keyPoint.x,y*pixelWidth - keyPoint.y};
        angle = fmodf(atan2f(gradient.y,gradient.x) + (2.0f*pi),2.0f*pi);//atan2f returns from -pi tp pi
        orientationHist[(int)floor(angle/rad10)] += getMagnitude(gradient)*expf(-((temp2.x*temp2.x)+(temp2.y*temp2.y))/weight);//pi/weight;
      }
    }
    // float3 convHelper = {orientationHist[35],orientationHist[0],orientationHist[1]};
    // for(int i = 0; i < 6; ++i){
    //  temp2.x = orientationHist[0];//need to hold on to this for id = 35 conv
    //  for(int id = 1; id < 36; ++id){
    //    orientationHist[id] = (convHelper.x+convHelper.y+convHelper.z)/3.0f;
    //    convHelper.x = convHelper.y;
    //    convHelper.y = convHelper.z;
    //    convHelper.z = (id < 35) ? orientationHist[id+1] : temp2.x;
    //    if(i == 5){
    //      if(orientationHist[id] > maxHist){
    //        maxHist = orientationHist[id];
    //      }
    //    }
    //  }
    // }
    for(int i = 0; i < 36; ++i){
      if(orientationHist[i] > maxHist) maxHist = orientationHist[i];
    }
    maxHist *= orientationThreshold;//% of max orientation value
    float2* bestMagWThetas = new float2[regNumOrient]();
    float2 tempMagWTheta = {0.0f,0.0f};
    for(int b = 0; b < 36; ++b){
      if(orientationHist[b] < maxHist ||
        (b > 0 && orientationHist[b] < orientationHist[b-1]) ||
        (b < 35 && orientationHist[b] < orientationHist[b+1]) ||
        (b == 0 && orientationHist[b] < orientationHist[35]) || 
        (b == 35 && orientationHist[b] < orientationHist[0]) ||
        (orientationHist[b] < bestMagWThetas[regNumOrient-1].x)){
        continue;
      } 

      tempMagWTheta.x = orientationHist[b];

      if(b == 0){
        tempMagWTheta.y = (orientationHist[35]-orientationHist[1])/(orientationHist[35]-(2.0f*orientationHist[0])+orientationHist[1]);
      }
      else if(b == 35){
        tempMagWTheta.y = (orientationHist[34]-orientationHist[0])/(orientationHist[34]-(2.0f*orientationHist[35])+orientationHist[0]);
      }
      else{
        tempMagWTheta.y = (orientationHist[b-1]-orientationHist[b+1])/(orientationHist[b-1]-(2.0f*orientationHist[b])+orientationHist[b+1]);
      }
      tempMagWTheta.y *= (pi/36.0f);
      tempMagWTheta.y += (b*rad10);
      tempMagWTheta.y = fmodf(tempMagWTheta.y + (2.0f*pi),2.0f*pi);

      for(int i = 0; i < regNumOrient; ++i){
        if(tempMagWTheta.x > bestMagWThetas[i].x){
          for(int ii = i; ii < regNumOrient; ++ii){
            temp2 = bestMagWThetas[ii];
            bestMagWThetas[ii] = tempMagWTheta;
            tempMagWTheta = temp2;
          }
        }
      }
    }
    for(int i = 0; i < regNumOrient; ++i){
      if(bestMagWThetas[i].x == 0.0f){
        thetaNumbers[globalID*regNumOrient + i] = -1;
        thetas[globalID*regNumOrient + i] = -FLT_MAX;
      }
      else{
        thetaNumbers[globalID*regNumOrient + i] = globalID;
        thetas[globalID*regNumOrient + i] = bestMagWThetas[i].y;
      }
    }
    delete[] bestMagWThetas;
  }
}

__global__ void ssrlcv::fillDescriptors(const unsigned long numFeatures, const unsigned int imageWidth, Feature<SIFT_Descriptor>* features,
    const float pixelWidth, const float lambda, const float windowWidth, const float* __restrict__ thetas,
    const int* __restrict__ keyPointAddresses, const float2* __restrict__ keyPointLocations, const float2* __restrict__ gradients){

  unsigned long blockId = blockIdx.y* gridDim.x+ blockIdx.x;
  if(blockId < numFeatures){
    __shared__ float normSq;
    __shared__ float bin_descriptors[4][4][8];
    bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] = 0.0f;
    float2 keyPoint = keyPointLocations[keyPointAddresses[blockId]];
    float theta = thetas[blockId];
    normSq = 0.0f;
    __syncthreads();
    /*
    FIRST DEFINE HOG GRID
    (x,y) = [(-8.5,-8.5),(8.5,8.5)]
      x' = xcos(theta) - ysin(theta) + feature.x
      y' = ycos(theta) + xsin(theta) + feature.y

    */

    float2 descriptorGridPoint = {0.0f,0.0f};

    float2 contribLoc = {0.0f,0.0f};
    float2 gradient = {0.0f,0.0f};
    float2 histLoc = {0.0f,0.0f};
    float temp = 0.0f;
    float binWidth = windowWidth/2.0f;
    float angle = 0.0f;

    float rad45 = pi/4.0f;
    for(float y = (float)threadIdx.y - windowWidth; y <= windowWidth; y+=(float)blockDim.y){
      if(threadIdx.z != 0) break;
      for(float x = (float)threadIdx.x - windowWidth; x <= windowWidth; x+=(float)blockDim.x){
        contribLoc = {(x*cosf(-theta)) + (y*sinf(-theta)),(-x*sinf(-theta)) + (y*cosf(-theta))};
        if(abs(contribLoc.x) > windowWidth || abs(contribLoc.y) > windowWidth) continue;
        //should interpolate to get proper gradient???
        gradient = gradients[llroundf(contribLoc.y + keyPoint.y)*imageWidth + llroundf(contribLoc.x + keyPoint.x)];
        descriptorGridPoint.x = getMagnitude(gradient)*expf(-((contribLoc.x*contribLoc.x)+(contribLoc.y*contribLoc.y))/(2.0f*windowWidth*windowWidth));///2.0f/pi/windowWidth/windowWidth; 
        descriptorGridPoint.y = fmodf(atan2f(gradient.y,gradient.x) - theta + (2.0f*pi),2.0f*pi);

        for(float nx = 0; nx < 4.0f; nx+=1.0f){
          for(float ny = 0; ny < 4.0f; ny+=1.0f){
            histLoc = {(nx*0.5f - 0.75f)*windowWidth,(ny*0.5f - 0.75f)*windowWidth};
            histLoc = {(histLoc.x*cosf(-theta)) + (histLoc.y*sinf(-theta)),(-histLoc.x*sinf(-theta)) + (histLoc.y*cosf(-theta))};
            histLoc = {abs(histLoc.x - contribLoc.x),abs(histLoc.y - contribLoc.y)};
            if(histLoc.x <= binWidth && histLoc.y <= binWidth){
              histLoc = histLoc/binWidth;
              for(int k = 0; k < 8; ++k){
                angle = abs(descriptorGridPoint.y-((float)k*rad45));
                if(angle < rad45){
                  angle /= rad45;
                  temp = (1.0f-histLoc.x)*(1.0f-histLoc.y)*(1.0f-angle)*descriptorGridPoint.x;
                  atomicAdd(&bin_descriptors[(int)nx][(int)ny][k],temp);
                }
              }
            }
          }
        }
      }
    }
    /*
    NORMALIZE
    */
    __syncthreads();
    atomicAdd(&normSq, bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]);
    __syncthreads();
    bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] /= sqrtf(normSq);
    if(bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] > 0.2f) bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] = 0.2f;
    __syncthreads();
    normSq = 0.0f;
    __syncthreads();
    atomicAdd(&normSq, bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]);
    __syncthreads();
    features[blockId].descriptor.values[(threadIdx.y*4 + threadIdx.x)*8 + threadIdx.z] = (unsigned char) roundf(255.0f*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]/sqrtf(normSq));
    if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
      features[blockId].descriptor.theta = theta;
      features[blockId].descriptor.sigma = 1.0f;
      features[blockId].loc = keyPoint*pixelWidth;//absolute location on image
    }
  }
}


__global__ void ssrlcv::checkKeyPoints(unsigned int numKeyPoints, unsigned int keyPointIndex, uint2 imageSize, float pixelWidth, float lambda, FeatureFactory::ScaleSpace::SSKeyPoint* keyPoints){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if(globalID < numKeyPoints){
    FeatureFactory::ScaleSpace::SSKeyPoint kp = keyPoints[globalID + keyPointIndex];
    float windowWidth = kp.sigma*lambda/pixelWidth;
    if((kp.loc.x - windowWidth) < 0.0f || 
    (kp.loc.y - windowWidth) < 0.0f || 
    (kp.loc.x + windowWidth) >= imageSize.x - 1||
    (kp.loc.y + windowWidth) >= imageSize.y - 1){
      keyPoints[globalID + keyPointIndex].discard = true;
    }
  }
}

/**
 * @brief This functions fills the descriptors.
 * 
 * @param numFeatures number of features
 * @param imageSize image size
 * @param features a pointer to a vector of SIFT feature descriptions
 * @param pixelWidth pixel width
 * @param lambda ???
 * @param keyPoints a pointer to the key points in the image
 * @param gradients ???
 * @return __global__ 
 */
__global__ void ssrlcv::fillDescriptors(unsigned int numFeatures, uint2 imageSize, Feature<SIFT_Descriptor>* features,
float pixelWidth, float lambda, FeatureFactory::ScaleSpace::SSKeyPoint* keyPoints, float2* gradients){
  unsigned long blockId = blockIdx.y* gridDim.x+ blockIdx.x;
  if(blockId < numFeatures){
    __shared__ float norm;
    norm = 0.0f;
    __shared__ float bin_descriptors[4][4][8];
    bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] = 0.0f;
    FeatureFactory::ScaleSpace::SSKeyPoint kp = keyPoints[blockId];
    float2 keyPoint = kp.loc;
    float windowWidth = ceil(kp.sigma*lambda/pixelWidth);
    float theta = kp.theta;
    __syncthreads();

    float2 descriptorGridPoint = {0.0f,0.0f};
    int imageWidth = imageSize.x;

    float2 contribLoc = {0.0f,0.0f};
    float2 gradient = {0.0f,0.0f};
    float2 histLoc = {0.0f,0.0f};
    float temp = 0.0f;
    float binWidth = windowWidth/2.0f;
    float angle = 0.0f;

    float rad45 = pi/4.0f;
    for(float y = (float)threadIdx.y - windowWidth; y <= windowWidth; y+=(float)blockDim.y){
      if(threadIdx.z != 0) break;
      for(float x = (float)threadIdx.x - windowWidth; x <= windowWidth; x+=(float)blockDim.x){
        contribLoc = {(x*cosf(-theta)) + (y*sinf(-theta)),(-x*sinf(-theta)) + (y*cosf(-theta))};
        if(abs(contribLoc.x) > windowWidth || abs(contribLoc.y) > windowWidth) continue;
        gradient = gradients[llroundf(contribLoc.y + keyPoint.y)*imageWidth + llroundf(contribLoc.x + keyPoint.x)];//this might need to be an interpolation
        descriptorGridPoint.x = getMagnitude(gradient)*expf(-((contribLoc.x*contribLoc.x)+(contribLoc.y*contribLoc.y))/(2.0f*windowWidth*windowWidth));//2.0f/pi/windowWidth/windowWidth; 
        descriptorGridPoint.y = fmodf(atan2f(gradient.y,gradient.x) - theta + (2.0f*pi),2.0f*pi);

        for(float nx = 0; nx < 4.0f; nx+=1.0f){
          for(float ny = 0; ny < 4.0f; ny+=1.0f){
            histLoc = {(nx*0.5f - 0.75f)*windowWidth,(ny*0.5f - 0.75f)*windowWidth};
            histLoc = {(histLoc.x*cosf(-theta)) + (histLoc.y*sinf(-theta)),(-histLoc.x*sinf(-theta)) + (histLoc.y*cosf(-theta))};
            histLoc = {abs(histLoc.x - contribLoc.x),abs(histLoc.y - contribLoc.y)};
            if(histLoc.x <= binWidth && histLoc.y <= binWidth){
              histLoc = histLoc/binWidth;
              for(int k = 0; k < 8; ++k){
                angle = abs(descriptorGridPoint.y-((float)k*rad45));
                if(angle < rad45){
                  angle /= rad45;
                  temp = (1.0f-histLoc.x)*(1.0f-histLoc.y)*(1.0f-angle)*descriptorGridPoint.x;
                  atomicAdd(&bin_descriptors[(int)nx][(int)ny][k],temp);
                }
              }
            }
          }
        }
      }
    }
    /*
    NORMALIZE
    */
    __syncthreads();
    atomicAdd(&norm, bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]);
    __syncthreads();
    bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] /= sqrtf(norm);
    if(bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] > 0.2f) bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z] = 0.2f;    
    __syncthreads();
    norm = 0.0f;    
    __syncthreads();
    atomicAdd(&norm,bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]);
    __syncthreads();
    features[blockId].descriptor.values[(threadIdx.y*4 + threadIdx.x)*8 + threadIdx.z] = (unsigned char) roundf(255.0f*bin_descriptors[threadIdx.x][threadIdx.y][threadIdx.z]/sqrtf(norm));
    if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
      features[blockId].descriptor.theta = kp.theta;
      features[blockId].descriptor.sigma = kp.sigma;
      features[blockId].loc = kp.loc*pixelWidth;//absolute location on image
    }
  }
} // fillDescriptors
