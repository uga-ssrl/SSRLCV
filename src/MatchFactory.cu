#include "hip/hip_runtime.h"
#include "MatchFactory.cuh"


/**
 * forward declarations of MatchFactories with descriptors that have distProtocol() implemented
 */
template class ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>;
template class ssrlcv::MatchFactory<ssrlcv::Window_3x3>;
template class ssrlcv::MatchFactory<ssrlcv::Window_9x9>;
template class ssrlcv::MatchFactory<ssrlcv::Window_15x15>;
template class ssrlcv::MatchFactory<ssrlcv::Window_25x25>;
template class ssrlcv::MatchFactory<ssrlcv::Window_31x31>;


template<typename T>
ssrlcv::MatchFactory<T>::MatchFactory(){
  this->relativeThreshold = 1.0f;
  this->absoluteThreshold = FLT_MAX;
  this->seedFeatures = nullptr;
}
template<typename T>
ssrlcv::MatchFactory<T>::MatchFactory(float relativeThreshold, float absoluteThreshold) :
relativeThreshold(relativeThreshold), absoluteThreshold(absoluteThreshold)
{
  this->seedFeatures = nullptr;
}
template<typename T>
void ssrlcv::MatchFactory<T>::setSeedFeatures(ssrlcv::ptr::value<Unity<Feature<T>>> seedFeatures){
  this->seedFeatures = seedFeatures;
}
template<typename T>
void ssrlcv::MatchFactory<T>::validateMatches(ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>> matches){
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);
  
  thrust::device_ptr<uint2_pair> needsValidating(matches->device.get());
  thrust::device_ptr<uint2_pair> new_end = thrust::remove_if(needsValidating,needsValidating+matches->size(),validate());
  hipDeviceSynchronize();
  CudaCheckError();
  int numMatchesLeft = new_end - needsValidating;
  if(numMatchesLeft == 0){
    logger.info<<"No valid matches found";
    matches.clear();
    return;
  }
  

  logger.info.printf("%d valid matches found out of %lu original matches",numMatchesLeft,matches->size());

  ssrlcv::ptr::device<uint2_pair> validatedMatches_device(numMatchesLeft);
  CudaSafeCall(hipMemcpy(validatedMatches_device.get(),matches->device.get(),numMatchesLeft*sizeof(uint2_pair),hipMemcpyDeviceToDevice));

  matches->setData(validatedMatches_device,numMatchesLeft,gpu);

  if(origin != gpu) matches->setMemoryState(origin);
}
template<typename T>
void ssrlcv::MatchFactory<T>::validateMatches(ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> matches){
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);
  
  thrust::device_ptr<Match> needsValidating(matches->device.get());
  thrust::device_ptr<Match> new_end = thrust::remove_if(needsValidating,needsValidating+matches->size(),validate());
  hipDeviceSynchronize();
  CudaCheckError();
  int numMatchesLeft = new_end - needsValidating;
  if(numMatchesLeft == 0){
    logger.info<<"No valid matches found";
    matches.clear();
    return;
  }
  

  logger.info.printf("%d valid matches found out of %lu original matches",numMatchesLeft,matches->size());

  ssrlcv::ptr::device<Match> validatedMatches_device(numMatchesLeft);
  CudaSafeCall(hipMemcpy(validatedMatches_device.get(),matches->device.get(),numMatchesLeft*sizeof(Match),hipMemcpyDeviceToDevice));

  matches->setData(validatedMatches_device,numMatchesLeft,gpu);

  if(origin != gpu) matches->setMemoryState(origin);
}
template<typename T>
void ssrlcv::MatchFactory<T>::validateMatches(ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::DMatch>> matches){
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);
  
  thrust::device_ptr<DMatch> needsValidating(matches->device.get());
  thrust::device_ptr<DMatch> new_end = thrust::remove_if(needsValidating,needsValidating+matches->size(),validate());
  hipDeviceSynchronize();
  CudaCheckError();
  int numMatchesLeft = new_end - needsValidating;
  if(numMatchesLeft == 0){
    logger.info<<"No valid matches found";
    matches.clear();
    return;
  }
  

  logger.info.printf("%d valid matches found out of %lu original matches",numMatchesLeft,matches->size());

  ssrlcv::ptr::device<DMatch> validatedMatches_device(numMatchesLeft);
  CudaSafeCall(hipMemcpy(validatedMatches_device.get(),matches->device.get(),numMatchesLeft*sizeof(DMatch),hipMemcpyDeviceToDevice));

  matches->setData(validatedMatches_device,numMatchesLeft,gpu);

  if(origin != gpu) matches->setMemoryState(origin);
}
template<typename T>
void ssrlcv::MatchFactory<T>::validateMatches(ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* matches){
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);

  FeatureMatch<T> *new_end = thrust::remove_if(matches->device.get(),matches->device.get()+matches->size(),validate());
  hipDeviceSynchronize();
  CudaCheckError();
  int numMatchesLeft = new_end - matches->device.get();
  if(numMatchesLeft == 0){
    logger.info<<"No valid matches found";
    delete matches;
    matches = nullptr;
    return;
  }
  

  logger.info.printf("%d valid matches found out of %lu original matches",numMatchesLeft,matches->size());

  ssrlcv::ptr::device<FeatureMatch<T>> validatedMatches_device(numMatchesLeft);
  CudaSafeCall(hipMemcpy(validatedMatches_device.get(),matches->device.get(),numMatchesLeft*sizeof(FeatureMatch<T>),hipMemcpyDeviceToDevice));

  matches->setData(validatedMatches_device,numMatchesLeft,gpu);

  if(origin != gpu) matches->setMemoryState(origin);

}
template<typename T>
void ssrlcv::MatchFactory<T>::refineMatches(ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::DMatch>> matches, float threshold){
  if(threshold == 0.0f){
    logger.err<<"ERROR illegal value used for threshold: 0.0";
    exit(-1);
  }
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);


  thrust::device_ptr<DMatch> needsCompacting(matches->device.get());
  thrust::device_ptr<DMatch> end = thrust::remove_if(needsCompacting, needsCompacting + matches->size(), match_dist_thresholder(threshold));
  unsigned int numElementsBelowThreshold = end - needsCompacting;
  if(numElementsBelowThreshold == 0){
    matches.clear();
    return;
  }

  logger.info.printf("%lu matches have been refined to %u matches using a cutoff of %f",matches->size(),numElementsBelowThreshold,threshold);

  ssrlcv::ptr::device<DMatch> compactedMatches_device(numElementsBelowThreshold);
  CudaSafeCall(hipMemcpy(compactedMatches_device.get(),matches->device.get(),numElementsBelowThreshold*sizeof(DMatch),hipMemcpyDeviceToDevice));

  matches->setData(compactedMatches_device,numElementsBelowThreshold,gpu);

  if(origin != gpu) matches->setMemoryState(origin);
}
template<typename T>
void ssrlcv::MatchFactory<T>::refineMatches(ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* matches, float threshold){
  if(threshold == 0.0f){
    logger.err<<"ERROR illegal value used for cutoff ratio: 0.0";
    exit(-1);
  }
  MemoryState origin = matches->getMemoryState();
  if(origin != gpu) matches->setMemoryState(gpu);

  thrust::device_ptr<FeatureMatch<T>> needsCompacting(matches->device.get());
  thrust::device_ptr<FeatureMatch<T>> end = thrust::remove_if(needsCompacting, needsCompacting + matches->size(), match_dist_thresholder(threshold));
  unsigned int numElementsBelowThreshold = end - needsCompacting;
  if(numElementsBelowThreshold == 0){
    delete matches;
    matches = nullptr;
    return;
  }

  logger.info.printf("%lu matches have been refined to %u matches using a cutoff of %f",matches->size(),numElementsBelowThreshold,threshold);

  ssrlcv::ptr::device<FeatureMatch<T>> compactedMatches_device(numElementsBelowThreshold);
  CudaSafeCall(hipMemcpy(compactedMatches_device.get(),matches->device.get(),numElementsBelowThreshold*sizeof(FeatureMatch<T>),hipMemcpyDeviceToDevice));

  matches->setData(compactedMatches_device,numElementsBelowThreshold,gpu);

  if(origin != gpu) matches->setMemoryState(origin);
}
template<typename T>
void ssrlcv::MatchFactory<T>::sortMatches(ssrlcv::ptr::value<ssrlcv::Unity<DMatch>> matches){
  if(matches->getFore() == gpu || matches->getFore() == both){
    thrust::device_ptr<DMatch> toSort(matches->device.get());
    thrust::sort(toSort, toSort + matches->size(),match_dist_comparator());
    matches->setFore(gpu);
    if(matches->getMemoryState() == both) matches->transferMemoryTo(cpu);
  }
  else if(matches->getFore() == cpu){
    unsigned long len = matches->size();
    // insertion sort
    // each match element is accessed with allMatches->host.get()[]
    unsigned long i = 0;
    unsigned long j = 0;
    ssrlcv::DMatch temp;
    while (i < len){
      j = i;
      while (j > 0 && matches->host.get()[j-1].distance > matches->host.get()[j].distance){
        temp = matches->host.get()[j];
        matches->host.get()[j] = matches->host.get()[j-1];
        matches->host.get()[j-1] = temp;
        j--;
      }
      i++;
    }
    if(matches->getMemoryState() == both) matches->transferMemoryTo(gpu);
  }
  else{
    logger.err<<"ERROR cannot perform sortMatches with matches->getMemoryState() = "<<std::to_string(matches->getMemoryState());
    exit(-1);
  }
}
template<typename T>
void ssrlcv::MatchFactory<T>::sortMatches(Unity<FeatureMatch<T>>* matches){
  if(matches->getFore() == gpu || matches->getFore() == both){
    thrust::sort(matches->device.get(), matches->device.get() + matches->size(),match_dist_comparator());
    matches->setFore(gpu);
    if(matches->getMemoryState() == both) matches->transferMemoryTo(cpu);
  }
  else if(matches->getFore() == cpu){
    unsigned long len = matches->size();
    // insertion sort
    // each match element is accessed with allMatches->host.get()[]
    unsigned long i = 0;
    unsigned long j = 0;
    ssrlcv::FeatureMatch<T> temp;
    while (i < len){
      j = i;
      while (j > 0 && matches->host.get()[j-1].distance > matches->host.get()[j].distance){
        temp = matches->host.get()[j];
        matches->host.get()[j] = matches->host.get()[j-1];
        matches->host.get()[j-1] = temp;
        j--;
      }
      i++;
    }
    if(matches->getMemoryState() == both) matches->transferMemoryTo(gpu);
  }
  else{
    logger.err<<"ERROR cannot perform sortMatches with matches->getMemoryState() = "<<std::to_string(matches->getMemoryState());
    exit(-1);
  }
}

template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::MatchFactory<T>::getRawMatches(ssrlcv::ptr::value<ssrlcv::Unity<DMatch>> matches){
  if(matches->getMemoryState() == gpu || matches->getFore() == gpu){
    ssrlcv::ptr::device<Match> rawMatches_device(matches->size());
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    void (*fp)(unsigned long, Match*, DMatch*) = &convertMatchToRaw;
    getFlatGridBlock(matches->size(),grid,block,fp);
    convertMatchToRaw<<<grid,block>>>(matches->size(),rawMatches_device.get(),matches->device.get());
    hipDeviceSynchronize();
    CudaCheckError();
    return ssrlcv::ptr::value<Unity<Match>>(rawMatches_device,matches->size(),gpu);
  }
  else{
    ssrlcv::ptr::host<Match> rawMatches_host(matches->size());
    for(int i = 0; i < matches->size(); ++i){
      for(int f = 0; f < 2; ++f){
        rawMatches_host.get()[i] = Match(matches->host.get()[i]);
      }
    }
    return ssrlcv::ptr::value<Unity<Match>>(rawMatches_host, matches->size(), cpu);
  }
}
template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::MatchFactory<T>::getRawMatches(Unity<FeatureMatch<T>>* matches){
  if(matches->getMemoryState() == gpu || matches->getFore() == gpu){
    ssrlcv::ptr::device<Match> rawMatches_device(matches->size());
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    void (*fp)(unsigned long, Match*, FeatureMatch<T>*) = &convertMatchToRaw;
    getFlatGridBlock(matches->size(),grid,block,fp);
    convertMatchToRaw<T><<<grid,block>>>(matches->size(),rawMatches_device.get(),matches->device.get());
    hipDeviceSynchronize();
    CudaCheckError();
    return ssrlcv::ptr::value<Unity<Match>>(rawMatches_device,matches->size(),gpu);
  }
  else{
    ssrlcv::ptr::host<Match> rawMatches_host(matches->size());
    for(int i = 0; i < matches->size(); ++i){
      for(int f = 0; f < 2; ++f){
        rawMatches_host.get()[i] = Match(matches->host.get()[i]);
      }
    }
    return ssrlcv::ptr::value<Unity<Match>>(rawMatches_host, matches->size(), cpu);
  }
}

/**
 * @brief This function computes the seed distances.
 * 
 * @tparam T 
 * @param features a pointer to a Unity data structure containing features of type T
 * @return a pointer to a Unity data structure containing the match distances
 */
template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<float>> ssrlcv::MatchFactory<T>::getSeedDistances(ssrlcv::ptr::value<Unity<Feature<T>>> features){
  MemoryState origin = features->getMemoryState();

  // set the memory state for seedFeatures and features to GPU
  if(this->seedFeatures->getMemoryState() != gpu) this->seedFeatures->setMemoryState(gpu);
  if(origin != gpu) features->setMemoryState(gpu);

  // the size of the feature vector is assigned to numPossibleMatches 
  unsigned int numPossibleMatches = features->size();

  ssrlcv::ptr::value<ssrlcv::Unity<float>> matchDistances = ssrlcv::ptr::value<ssrlcv::Unity<float>>(nullptr, numPossibleMatches,gpu);

  // initilize grid and block dimensions
  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matchDistances->size(),grid);

  clock_t timer = clock();

  // call the kernel function getSeedMatchDistances
  getSeedMatchDistances<T><<<grid, block>>>(features->size(),features->device.get(),this->seedFeatures->size(),
    this->seedFeatures->device.get(),matchDistances->device.get());

  hipDeviceSynchronize();
  CudaCheckError();

  logger.info.printf("seed match distances computed in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin != gpu) features->setMemoryState(origin);
  
  return matchDistances;
} // getSeedDistances

template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::MatchFactory<T>::generateMatches(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<Match> matches_device( numPossibleMatches);
  ssrlcv::ptr::value<ssrlcv::Unity<Match>> matches = ssrlcv::ptr::value<ssrlcv::Unity<Match>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
  
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::MatchFactory<T>::generateMatchesConstrained(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, float epsilon, float fundamental[3][3], ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<Match> matches_device( numPossibleMatches);

  ssrlcv::ptr::value<ssrlcv::Unity<Match>> matches = ssrlcv::ptr::value<ssrlcv::Unity<Match>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  ssrlcv::ptr::device<float> fundamental_device(9);
  CudaSafeCall(hipMemcpy(fundamental_device.get(),fundamental,9*sizeof(float),hipMemcpyHostToDevice));

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon,fundamental_device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),epsilon,fundamental_device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }

  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
}


template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::DMatch>> ssrlcv::MatchFactory<T>::generateDistanceMatches(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::value<ssrlcv::Unity<DMatch>> matches = ssrlcv::ptr::value<ssrlcv::Unity<DMatch>>(nullptr, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
} // generateDistanceMatches

template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::DMatch>> ssrlcv::MatchFactory<T>::generateDistanceMatchesKDTree(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::KDTree kdtree, ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances) {

  // transfer query points to GPU
  MemoryState q_origin = queryFeatures->getMemoryState();
  if(q_origin != gpu) queryFeatures->setMemoryState(gpu);

  // transfer KD-Tree to GPU
  ssrlcv::ptr::device<ssrlcv::KDTree> d_kdtree(1);
  CudaSafeCall(hipMemcpy(d_kdtree.get(),&kdtree,sizeof(kdtree),hipMemcpyHostToDevice));
   
  // transfer KD-Tree nodes to GPU
  thrust::device_vector<typename KDTree::Node> d_nodes = kdtree.nodes;
  typename KDTree::Node* pd_nodes = thrust::raw_pointer_cast(d_nodes.data());

  // transfer KD-Tree points to GPU
  ssrlcv::ptr::value<ssrlcv::Unity<Feature<ssrlcv::SIFT_Descriptor>>> d_points = kdtree.points; 
  MemoryState t_origin = d_points->getMemoryState();
  if(t_origin != gpu) d_points->setMemoryState(gpu); 

  // array to hold the matched pairs
  unsigned int numPossibleMatches = queryFeatures->size();
  ssrlcv::ptr::value<ssrlcv::Unity<DMatch>> matches = ssrlcv::ptr::value<ssrlcv::Unity<DMatch>>(nullptr, numPossibleMatches, gpu);

  // grid and block initilization
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  void (*ptr)(unsigned int, unsigned long, Feature<ssrlcv::SIFT_Descriptor>*, unsigned int, KDTree*,
  typename KDTree::Node*, Feature<ssrlcv::SIFT_Descriptor>*, DMatch*, float) = &matchFeaturesKDTree;
  getFlatGridBlock(queryFeatures->size(), grid, block, ptr);

  clock_t timer = clock();
  
  if (seedDistances == nullptr) {
    matchFeaturesKDTree<<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(), 
    target->id, d_kdtree.get(), pd_nodes, d_points->device.get(), matches->device.get(), this->absoluteThreshold);
  } else if (seedDistances->size() != queryFeatures->size()) {
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures"<<"\n";
    exit(-1);
  } else {
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesKDTree<<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(), 
    target->id, d_kdtree.get(), pd_nodes, d_points->device.get(), matches->device.get(), seedDistances->device.get(),
    this->relativeThreshold, this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin); 
  }
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);
  printf("\n\ndone in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);
  if(q_origin != gpu) queryFeatures->setMemoryState(q_origin);
  if(t_origin != gpu) kdtree.points->setMemoryState(t_origin);

  return matches;
} // generateDistanceMatchesKDTree

template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::DMatch>>ssrlcv::MatchFactory<T>:: generateDistanceMatchesConstrained(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, float epsilon, float fundamental[3][3], ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<DMatch> matches_device( numPossibleMatches);

  ssrlcv::ptr::value<ssrlcv::Unity<DMatch>> matches = ssrlcv::ptr::value<ssrlcv::Unity<DMatch>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  ssrlcv::ptr::device<float> fundamental_device(9);
  CudaSafeCall(hipMemcpy(fundamental_device.get(),fundamental,9*sizeof(float),hipMemcpyHostToDevice));

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon, fundamental_device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon, fundamental_device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
} // generateDistanceMatchesConstrained


template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateFeatureMatches(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<T>>> queryFeatures,
ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<T>>> targetFeatures, ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){

  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};
  
  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<FeatureMatch<T>> matches_device( numPossibleMatches);

  Unity<FeatureMatch<T>>* matches = new Unity<FeatureMatch<T>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
    
  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateFeatureMatchesConstrained(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<T>>> queryFeatures,
ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<T>>> targetFeatures, float epsilon, float fundamental[3][3], ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){

  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};
  
  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<FeatureMatch<T>> matches_device( numPossibleMatches);

  Unity<FeatureMatch<T>>* matches = new Unity<FeatureMatch<T>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  ssrlcv::ptr::device<float> fundamental_device(9);
  CudaSafeCall(hipMemcpy(fundamental_device.get(),fundamental,9*sizeof(float),hipMemcpyHostToDevice));

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon, fundamental_device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon, fundamental_device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;

}



template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::uint2_pair>> ssrlcv::MatchFactory<T>::generateMatchesIndexOnly(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<uint2_pair> matches_device( numPossibleMatches);
  ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>> matches = ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }
  
  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::uint2_pair>> ssrlcv::MatchFactory<T>::generateMatchesConstrainedIndexOnly(ssrlcv::ptr::value<ssrlcv::Image> query, ssrlcv::ptr::value<Unity<Feature<T>>> queryFeatures, ssrlcv::ptr::value<ssrlcv::Image> target, ssrlcv::ptr::value<Unity<Feature<T>>> targetFeatures, float epsilon, float fundamental[3][3], ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances){
  MemoryState origin[2] = {queryFeatures->getMemoryState(), targetFeatures->getMemoryState()};

  if(origin[0] != gpu) queryFeatures->setMemoryState(gpu);
  if(origin[1] != gpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->size();

  ssrlcv::ptr::device<uint2_pair> matches_device( numPossibleMatches);

  ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>> matches = ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {32,1,1};//IMPROVE
  getGrid(matches->size(),grid);

  ssrlcv::ptr::device<float> fundamental_device(9);
  CudaSafeCall(hipMemcpy(fundamental_device.get(),fundamental,9*sizeof(float),hipMemcpyHostToDevice));

  clock_t timer = clock();

  if(seedDistances == nullptr){
    matchFeaturesConstrained<<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(), epsilon,fundamental_device.get(),this->absoluteThreshold);
  }
  else if(seedDistances->size() != queryFeatures->size()){
    logger.err<<"ERROR: seedDistances should have come from matching a seed image to queryFeatures";
    exit(-1);
  }
  else{
    MemoryState seedOrigin = seedDistances->getMemoryState();
    if(seedOrigin != gpu) seedDistances->setMemoryState(gpu);
    matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->size(), queryFeatures->device.get(),
    target->id, targetFeatures->size(), targetFeatures->device.get(), matches->device.get(),epsilon,fundamental_device.get(),seedDistances->device.get(),
    this->relativeThreshold,this->absoluteThreshold);
    if(seedOrigin != gpu) seedDistances->setMemoryState(seedOrigin);
  }

  hipDeviceSynchronize();
  CudaCheckError();

  this->validateMatches(matches);

  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetFeatures->setMemoryState(origin[1]);

  return matches;
}


template<typename T>
ssrlcv::MatchSet ssrlcv::MatchFactory<T>::generateMatchesExaustive(std::vector<ssrlcv::ptr::value<ssrlcv::Image>> images, std::vector<ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Feature<T>>>> features, bool ordered, float estimatedOverlap){
  MatchSet matchSet;
  matchSet.keyPoints = nullptr;
  matchSet.matches = nullptr;
  if(estimatedOverlap == 0){
    logger.warn<<"WARNING: estimated overlap fraction of 0.0f requires unordered match interpolation";
  }
  std::vector<ssrlcv::ptr::value<ssrlcv::Image>>::iterator query = images.begin();
  std::vector<ssrlcv::ptr::value<ssrlcv::Image>>::iterator target = query + 1;
  typename std::vector<ssrlcv::ptr::value<Unity<Feature<T>>>>::iterator features_query = features.begin();
  typename std::vector<ssrlcv::ptr::value<Unity<Feature<T>>>>::iterator features_target = features_query + 1;
  std::vector<ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>>> matchIndices;
  ssrlcv::ptr::value<ssrlcv::Unity<float>> seedDistances;
  unsigned long long totalMatches = 0;
  int i = 0;
  logger.info<<"matching images";
  for(int i = 0; query != images.end() - 1; ++query, ++features_query){
    if(this->seedFeatures != nullptr) seedDistances = this->getSeedDistances(*features_query);
    for(target = query + 1,features_target = features_query + 1; target != images.end(); ++target, ++features_target){
      if(ordered && estimatedOverlap > 0.0f && ++i*(1-estimatedOverlap) > 1.0f) continue; //based off linear images
      //now match
      matchIndices.push_back(this->generateMatchesIndexOnly(*query,*features_query,*target,*features_target,seedDistances));
      totalMatches += matchIndices[i++]->size();
    }
  }
  if(totalMatches == 0){
    logger.err<<"There were no matches found in the set of images, likely due to unreasonable threshold";
    logger.err<<"exiting...";
    exit(0);
  }
  logger.info<<"prepping match interpolation on cpu";
  //required connections to make a match?
  std::vector<uint2>** adjacencyList = new std::vector<uint2>*[images.size() - 1];

  i = 0;
  adjacencyList[0] = new std::vector<uint2>[features[0]->size()];
  logger.info<<"building adjacency list";
  for(auto m = matchIndices.begin(); m != matchIndices.end(); ++m){
    ssrlcv::ptr::value<ssrlcv::Unity<uint2_pair>> currentMatches = *m;
    if(currentMatches->getMemoryState() != cpu) currentMatches->setMemoryState(cpu);
    if(currentMatches->host.get()[0].a.x != i){
      i++;
      adjacencyList[i] = new std::vector<uint2>[features[i]->size()];
    }
    for(int p = 0; p < currentMatches->size(); ++p){
      uint2_pair* currentPair = &currentMatches->host.get()[p];
      adjacencyList[currentPair->a.x][currentPair->a.y].push_back(currentPair->b); 
    }
  }
  MemoryState* origin = new MemoryState[images.size()];
  std::vector<std::vector<uint2>> multiMatch_vec;
  bool badMatch = false;
  logger.info<<"deriving matches from adjacency";
  for(i = 0; i < images.size() - 1; ++i){
    origin[i] = features[i]->getMemoryState();
    if(origin[i] != cpu) features[i]->setMemoryState(cpu);
    for(int f = 0; i < images.size() - 2 && f < features[i]->size(); ++f){
      std::vector<uint2>* adj = &adjacencyList[i][f];
      if(!adj->size()) continue;
      badMatch = false;
      std::vector<uint2>* prev_adj = adj;
      std::vector<uint2>* next_adj = nullptr;  
      while(true){
        if(prev_adj->begin()->x == images.size() - 1) break;
        next_adj = &adjacencyList[prev_adj->begin()->x][prev_adj->begin()->y];
        if(!next_adj->size()) break;
        std::vector<uint2> intersection;
        std::set_intersection(prev_adj->begin(),prev_adj->end(),next_adj->begin(),next_adj->end(),std::back_inserter(intersection));
        if(intersection.size() != next_adj->size()){
          badMatch = true;
          break;
        }
        else if(next_adj->size() == 1) break;
        else{
          prev_adj = next_adj;
        }
      } 
      if(badMatch) adj->clear();
      else{
        std::vector<uint2> match;
        match.push_back({(unsigned int)i,(unsigned int)f});
        match.insert(match.end(),adjacencyList[i][f].begin(),adjacencyList[i][f].end());
        multiMatch_vec.push_back(match);
        for(auto m = adj->begin(); m != adj->end() - 1; ++m){
          if(m->x == images.size() - 1) break;
          next_adj = &adjacencyList[m->x][m->y];
          next_adj->clear();
        }
      } 
    }
    delete[] adjacencyList[i];
  }
  delete[] adjacencyList;
  logger.info.printf("total matches found in set = %d", multiMatch_vec.size());
  matchSet.matches = ssrlcv::ptr::value<ssrlcv::Unity<MultiMatch>>(nullptr,multiMatch_vec.size(),cpu);
  std::vector<KeyPoint> kp_vec;
  i = 0;
  int index = 0;
  for(auto m = multiMatch_vec.begin(); m != multiMatch_vec.end(); ++m){
    matchSet.matches->host.get()[i++] = {(unsigned int)m->size(),index};
    index += m->size();
    for(auto kp = m->begin(); kp != m->end(); ++kp){
      kp_vec.push_back({(int)kp->x,features[kp->x]->host.get()[kp->y].loc});
    }
  }
  matchSet.keyPoints = ssrlcv::ptr::value<ssrlcv::Unity<KeyPoint>>(nullptr,kp_vec.size(),gpu);
  CudaSafeCall(hipMemcpy(matchSet.keyPoints->device.get(),&kp_vec[0],kp_vec.size()*sizeof(KeyPoint),hipMemcpyHostToDevice));
  for(int i = 0; i < images.size() - 1; ++i){
    if(origin[i] != cpu) features[i]->setMemoryState(origin[i]);
  }
  delete[] origin;
  //1:2,1:3,1:4,1:5,2:3,2:4,2:5,3:4,3:5,4:5
  return matchSet;

}


ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::generateDiparityMatches(uint2 querySize, ssrlcv::ptr::value<ssrlcv::Unity<unsigned char>> queryPixels, uint2 targetSize, ssrlcv::ptr::value<ssrlcv::Unity<unsigned char>> targetPixels, 
  float fundamental[3][3], unsigned int maxDisparity,unsigned int windowSize, Direction direction){
  if(direction != right && direction != left && direction != undefined){
    logger.err<<"ERROR: unsupported search direction for disparity matching";
    exit(-1);
  }
  if(maxDisparity > querySize.x){
    logger.err<<"Max disparity cannot be larger than image size";
    exit(-1);
  }
  logger.info.printf("running disparity matching on parallel images");
  logger.info.printf("\timage[0] = %ux%u",querySize.x,querySize.y);
  logger.info.printf("\timage[1] = %ux%u",targetSize.x,targetSize.y);
  logger.info.printf("\tmaxDisparity = %u",maxDisparity);
  logger.info.printf("\twindow size = %ux%u",windowSize,windowSize);

  if(windowSize == 0 || windowSize % 2 == 0 || windowSize > 31){
    logger.err<<"ERROR window size for disparity matching must be greater than 0, less than 31 and odd";
    exit(-1);
  }

  MemoryState origin[2] = {queryPixels->getMemoryState(), targetPixels->getMemoryState()};

  if(origin[0] != gpu) queryPixels->setMemoryState(gpu);
  if(origin[1] != gpu) targetPixels->setMemoryState(gpu);
  
  uint2 minimizedSize = {querySize.x-windowSize-1,querySize.y-windowSize-1};

  unsigned int numPossibleMatches = minimizedSize.x*minimizedSize.y;

  ssrlcv::ptr::device<Match> matches_device( numPossibleMatches);

  ssrlcv::ptr::value<ssrlcv::Unity<Match>> matches = ssrlcv::ptr::value<ssrlcv::Unity<Match>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {windowSize,windowSize,1};//NOTE some devices will not be able to handle large numbers here
  checkDims(grid,block);
  getGrid(numPossibleMatches,grid);

  bool parallel = true;
  for(int x = 0; x < 3 && parallel; ++x){
    for(int y = 0; y < 3; ++y){
      if((x == 2 && y == 1 && fundamental[y][x] == -1.0f) || (x == 1 && y == 2 && fundamental[y][x] == 1.0f)) continue;
      if(fundamental[y][x] != 0.0f){
        parallel = false;
        break;
      }
    }
  }

  clock_t timer = clock();

  if(!parallel){
    ssrlcv::ptr::device<float> fundamental_device(9);
    CudaSafeCall(hipMemcpy(fundamental_device.get(),fundamental,9*sizeof(float),hipMemcpyHostToDevice));
    disparityMatching<<<grid, block>>>(querySize,queryPixels->device.get(),targetSize,targetPixels->device.get(),fundamental_device.get(),matches->device.get(),maxDisparity,direction);
  }
  else{
    disparityScanMatching<<<grid,block>>>(querySize,queryPixels->device.get(),targetSize,targetPixels->device.get(),matches->device.get(),maxDisparity,direction);
  }
  
  hipDeviceSynchronize();
  CudaCheckError();  
  logger.info.printf("done in %f seconds.",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != gpu) queryPixels->setMemoryState(origin[0]);
  if(origin[1] != gpu) targetPixels->setMemoryState(origin[1]);

  thrust::device_ptr<Match> needsValidating(matches->device.get());
  thrust::device_ptr<Match> new_end = thrust::remove_if(needsValidating,needsValidating+matches->size(),validate());
  hipDeviceSynchronize();
  CudaCheckError();
  int numMatchesLeft = new_end - needsValidating;
  if(numMatchesLeft == 0){
    logger.info<<"No valid matches found";
    matches.clear();
  }
  else{
    logger.info.printf("%d valid matches found out of %lu original matches",numMatchesLeft,matches->size());
    ssrlcv::ptr::device<Match> validatedMatches_device(numMatchesLeft);
    CudaSafeCall(hipMemcpy(validatedMatches_device.get(),matches->device.get(),numMatchesLeft*sizeof(Match),hipMemcpyDeviceToDevice));
    matches->setData(validatedMatches_device,numMatchesLeft,gpu);
  }
  //returning based on origin of query pixels
  if(origin[0] != gpu) matches->setMemoryState(origin[0]);
  return matches;
}


void ssrlcv::writeMatchFile(ssrlcv::ptr::value<ssrlcv::Unity<Match>> matches, std::string pathToFile, bool binary){
  MemoryState origin = matches->getMemoryState();
  if(matches->getFore() == gpu) matches->transferMemoryTo(cpu);
  if(binary){
    std::ofstream matchstream(pathToFile,std::ios_base::binary);
    if(matchstream.is_open()){
      for(int i = 0; i < matches->size(); ++i){
        matchstream.write((char*)&matches->host.get()[i].keyPoints[0].loc,2*sizeof(float));
        matchstream.write((char*)&matches->host.get()[i].keyPoints[1].loc,2*sizeof(float));
      }
    }
    else{
      logger.err<<"ERROR: cannot write "<<pathToFile;
    }
    matchstream.close();
  }
  else{
    std::ofstream matchstream(pathToFile);
    if(matchstream.is_open()){
      std::string line;
      for(int i = 0; i < matches->size(); ++i){
        line = std::to_string(matches->host.get()[i].keyPoints[0].loc.x) + ",";
        line += std::to_string(matches->host.get()[i].keyPoints[0].loc.y) + ",";
        line += std::to_string(matches->host.get()[i].keyPoints[1].loc.x) + ",";
        line += std::to_string(matches->host.get()[i].keyPoints[1].loc.y) + "\n";
        matchstream << line;
      }
      matchstream.close();
    }
    else{
      logger.err<<"ERROR: cannot write match files";
      exit(-1);
    }
  }
  logger.info << pathToFile + " has been written";
  if(origin != matches->getMemoryState()) matches->setMemoryState(origin);
}
void ssrlcv::writeMatchFile(MatchSet multiview_matches, std::string pathToFile, bool binary){
  ssrlcv::ptr::value<ssrlcv::Unity<MultiMatch>> matches = multiview_matches.matches;
  ssrlcv::ptr::value<ssrlcv::Unity<KeyPoint>> keyPoints = multiview_matches.keyPoints;
  MemoryState origin[2] = {matches->getMemoryState(),keyPoints->getMemoryState()};
  if(origin[0] != cpu) matches->setMemoryState(cpu);
  if(origin[1] != cpu) keyPoints->setMemoryState(cpu);

  std::ofstream matchstream(pathToFile);
  if(matchstream.is_open()){
    std::string line;
    for(int i = 0; i < matches->size(); ++i){
      line = std::to_string(matches->host.get()[i].numKeyPoints) + ",";
      for(int kp = matches->host.get()[i].index; kp < matches->host.get()[i].index + matches->host.get()[i].numKeyPoints; ++kp){
        line += std::to_string(keyPoints->host.get()[kp].parentId) + ",";
        line += std::to_string(keyPoints->host.get()[kp].loc.x) + ",";
        line += std::to_string(keyPoints->host.get()[kp].loc.y) + ",";
      }
      line += "\n";
      matchstream << line;
    }
    matchstream.close();
  }
  else{
    logger.err<<"ERROR: cannot write match files";
    exit(-1);
  }
  
  logger.info << pathToFile + " has been written";
  if(origin[0] != cpu) matches->setMemoryState(origin[0]);
  if(origin[1] != cpu) keyPoints->setMemoryState(origin[1]);
}

//NOTE currently only capable of reading in pairwise match files
ssrlcv::ptr::value<ssrlcv::Unity<ssrlcv::Match>> ssrlcv::readMatchFile(std::string pathToFile){
  std::ifstream matchstream(pathToFile);
  std::vector<Match> match_vec;
  if(matchstream.is_open()){
    std::string line;
    std::string item;
    getline(matchstream,line);//calibration parameters
    while(getline(matchstream,line)){
      std::istringstream s(line);
      Match match = Match();
      match.keyPoints[0].parentId = 0;
      match.keyPoints[1].parentId = 1;
      getline(s,item,',');
      match.keyPoints[0].loc.x = std::stof(item);
      getline(s,item,',');
      match.keyPoints[0].loc.y = std::stof(item);
      getline(s,item,',');
      match.keyPoints[1].loc.x = std::stof(item);
      getline(s,item,',');
      match.keyPoints[1].loc.y = std::stof(item);
      match_vec.push_back(match);
    }
  }
  logger.info.printf("%d matches have been read.", match_vec.size());
  ssrlcv::ptr::value<ssrlcv::Unity<Match>> matches = ssrlcv::ptr::value<ssrlcv::Unity<Match>>(nullptr,match_vec.size(),cpu);
  std::memcpy(matches->host.get(),&match_vec[0],match_vec.size()*sizeof(Match));
  return matches;
}


/*
CUDA implementations
*/

__host__ __device__ __forceinline__ float ssrlcv::sum(const float3 &a){
  return a.x + a.y + a.z;
}
__host__ __device__ __forceinline__ float ssrlcv::square(const float &a){
  return a*a;
}
__device__ __forceinline__ float ssrlcv::atomicMinFloat (float * addr, float value) {
  float old;
  old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
  return old;
}

/*
  diparity matching kernels
*/

__global__ void ssrlcv::disparityMatching(uint2 querySize, unsigned char* pixelsQuery, uint2 targetSize, unsigned char* pixelsTarget, float* fundamental, Match* matches, unsigned int maxDisparity, Direction direction){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  uint2 minimizedSize = {querySize.x-blockDim.x-1,querySize.y-blockDim.x-1};
  if(blockId < minimizedSize.x*minimizedSize.y){
    uint2 loc = {blockId%minimizedSize.x + (blockDim.x/2),blockId/minimizedSize.x + (blockDim.y/2)};
    uint2 threadLoc = {threadIdx.x - (blockDim.x/2),threadIdx.y - (blockDim.y/2)};
    __shared__ int3 matchInfo;
    __shared__ int currentDist;
    __shared__ float3 epipolar;
    __shared__ int2 searchLoc;
    __shared__ int stop;
    stop = maxDisparity; 
    int stride = 1;
    if(threadIdx.x + threadIdx.y == 0){
      matchInfo = {-1,-1,INT_MAX};
      currentDist = 0;
      epipolar.x = (fundamental[0]*loc.x) + (fundamental[1]*loc.y) + fundamental[2];
      epipolar.y = (fundamental[3]*loc.x) + (fundamental[4]*loc.y) + fundamental[5];
      epipolar.z = (fundamental[6]*loc.x) + (fundamental[7]*loc.y) + fundamental[8];
      if(direction == right){
        searchLoc.x = loc.x;
        stop -= querySize.x - ((int)maxDisparity + (int)loc.x);
      }
      else if(direction == left){
        stride = -1;
        searchLoc.x = loc.x;
        stop += loc.x - (int)maxDisparity;
      }
      else{
        searchLoc.x = loc.x - ((int)maxDisparity/2);
        if(searchLoc.x < 0){
          searchLoc.x = 0;
        }
      }
      searchLoc.y = (int)floor(-1*((epipolar.x*searchLoc.x) + epipolar.z)/epipolar.y);
    }  
    __syncthreads();

    int threadPixel = pixelsQuery[(loc.y + threadLoc.y)*querySize.x + loc.x + threadLoc.x];
    for(int i = 0; i < stop; ++i){
      atomicAdd(&currentDist,abs(threadPixel-(int)pixelsTarget[(searchLoc.y+threadLoc.y)*targetSize.x + searchLoc.x + threadLoc.x]));
      __syncthreads();
      if(threadIdx.x + threadIdx.y == 0){
        if(currentDist < matchInfo.z){
          matchInfo = {searchLoc.x,searchLoc.y,currentDist};
        }
        searchLoc.x+=stride;
        searchLoc.y = (int)floor(-1*((epipolar.x*searchLoc.x) + epipolar.z)/epipolar.y);
        currentDist = 0;
      } 
      __syncthreads();
    }

    Match match;
    if(matchInfo.x == -1){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = {(float)loc.x + threadLoc.x,(float)loc.y + threadLoc.y};
      match.keyPoints[1].loc = {(float)matchInfo.x + threadLoc.x,(float)loc.y + threadLoc.y};
      match.keyPoints[0].parentId = 0;
      match.keyPoints[1].parentId = 1;
    }
    matches[(loc.y+threadLoc.y)*minimizedSize.x + loc.x + threadLoc.x] = match;
  }
}
__global__ void ssrlcv::disparityScanMatching(uint2 querySize, unsigned char* pixelsQuery, uint2 targetSize, unsigned char* pixelsTarget, Match* matches, unsigned int maxDisparity, Direction direction){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  uint2 minimizedSize = {querySize.x-blockDim.x-1,querySize.y-blockDim.x-1};
  if(blockId < minimizedSize.x*minimizedSize.y){
    uint2 loc = {blockId%minimizedSize.x + (blockDim.x/2), blockId/minimizedSize.x + (blockDim.y/2)};
    uint2 threadLoc = {threadIdx.x - (blockDim.x/2),threadIdx.y - (blockDim.y/2)};
    __shared__ int2 matchInfo;
    __shared__ int currentDist;
    __shared__ int searchX;
    __shared__ int stop;
    stop = maxDisparity;
    int stride = 1;
    if(threadIdx.x + threadIdx.y == 0){
      matchInfo = {-1,INT_MAX};
      currentDist = 0;
      if(direction == right){
        searchX = loc.x;
        if(stop + loc.x > targetSize.x){
          stop = targetSize.x - loc.x;
        }
      }
      else if(direction == left){
        stride = -1;
        searchX = loc.x;
        if((int)loc.x - stop < 0){
          stop = loc.x;
        }
      }
      else{
        searchX = loc.x - ((int)maxDisparity/2);
        if(searchX < 0){
          searchX = 0;
        }
      }
    }
    __syncthreads();
    int threadPixel = pixelsQuery[(loc.y + threadLoc.y)*querySize.x + loc.x + threadLoc.x];
    int indexHelper = (loc.y + threadLoc.y)*targetSize.x;
    for(int i = 0; i < stop; ++i){
      atomicAdd(&currentDist,abs(threadPixel - (int)pixelsTarget[indexHelper + searchX + threadLoc.x]));
      __syncthreads();
      if(threadIdx.x + threadIdx.y == 0){
        if(currentDist < matchInfo.y){
          matchInfo = {searchX,currentDist};
        }
        searchX+=stride;
        currentDist = 0;
      }
      __syncthreads();
    }
    
    Match match;
    if(matchInfo.x == -1){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = {(float)loc.x + threadLoc.x,(float)loc.y + threadLoc.y};
      match.keyPoints[1].loc = {(float)matchInfo.x + threadLoc.x,(float)loc.y + threadLoc.y};
      match.keyPoints[0].parentId = 0;
      match.keyPoints[1].parentId = 1;
    }
    matches[(loc.y+threadLoc.y)*minimizedSize.x + loc.x + threadLoc.x] = match;
  }
}


/*
  Matching kernels
*/

/**
 * @brief This function computes the seed match distances.
 * 
 * @tparam T 
 * @param numFeaturesQuery the number of features to query 
 * @param featuresQuery a ptr to the feature vector to be queried
 * @param numSeedFeatures the number of seed features
 * @param seedFeatures a ptr to the feature vector containing the seed features
 * @param matchDistances a ptr to the match distances
 * @return __global__ 
 */
template<typename T>
__global__ void ssrlcv::getSeedMatchDistances(unsigned long numFeaturesQuery, Feature<T>* featuresQuery, unsigned long numSeedFeatures,
Feature<T>* seedFeatures, float* matchDistances){
  // define the block ID
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ float localDist[32];
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numSeedFeatures_reg = numSeedFeatures;
    for(int f = threadIdx.x; f < numSeedFeatures_reg; f += 32){
      currentDist = feature.descriptor.distProtocol(seedFeatures[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
      }
    } // for
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
      }
    } // if
    matchDistances[blockId] = currentDist;
  } // if
} // getSeedMatchDistances


template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    if(currentDist >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches, float epsilon, float* fundamental, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    if(currentDist >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches, float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    if(currentDist >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(currentDist/nearestSeed > relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches, float epsilon, float* fundamental, float* seedDistances, 
float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    if(currentDist >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(currentDist/nearestSeed > relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}


template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches, float epsilon, float* fundamental, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches, 
float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(match.distance/nearestSeed > relativeThreshold*relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches, float epsilon, float* fundamental, 
float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(match.distance/nearestSeed > relativeThreshold*relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}
//template<typename T>
__global__ void ssrlcv::matchFeaturesKDTree(unsigned int queryImageID, unsigned long numFeaturesQuery, ssrlcv::Feature<ssrlcv::SIFT_Descriptor>* featuresQuery,
unsigned int targetImageID, ssrlcv::KDTree* kdtree, typename ssrlcv::KDTree::Node* nodes, ssrlcv::Feature<ssrlcv::SIFT_Descriptor>* featuresTree,
ssrlcv::DMatch* matches, float absoluteThreshold) {
  
  unsigned int globalThreadID = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; // 2D grid of 1D blocks 
  if (globalThreadID < numFeaturesQuery) { 
    Feature<ssrlcv::SIFT_Descriptor> feature = featuresQuery[globalThreadID]; 
    __syncthreads(); 
    DMatch match;
    int emax = 100; // at most, search 100 leaf nodes
    match = findNearest(kdtree, nodes, featuresTree, feature, queryImageID, targetImageID, emax, absoluteThreshold); // search function 
    __syncthreads();
    matches[globalThreadID] = match;
  } 

} // matchFeaturesKDTree

//template<typename T>
__global__ void ssrlcv::matchFeaturesKDTree(unsigned int queryImageID, unsigned long numFeaturesQuery, ssrlcv::Feature<ssrlcv::SIFT_Descriptor>* featuresQuery,
unsigned int targetImageID, ssrlcv::KDTree* kdtree, typename ssrlcv::KDTree::Node* nodes, ssrlcv::Feature<ssrlcv::SIFT_Descriptor>* featuresTree,
ssrlcv::DMatch* matches, float* seedDistances, float relativeThreshold, float absoluteThreshold) {
  
  unsigned int globalThreadID = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; // 2D grid of 1D blocks
  
  if (globalThreadID < numFeaturesQuery) { 
    Feature<ssrlcv::SIFT_Descriptor> feature = featuresQuery[globalThreadID];
    float nearestSeed = seedDistances[globalThreadID];
    __syncthreads();
    
    DMatch match;
    int emax = 100; // at most, search 100 leaf nodes
    match = findNearest(kdtree, nodes, featuresTree, feature, queryImageID, targetImageID, emax, relativeThreshold, absoluteThreshold, nearestSeed); 
    __syncthreads();
    matches[globalThreadID] = match;
  } 

} // matchFeaturesKDTree

template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;    
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.descriptors[0] = feature.descriptor;
      match.descriptors[1] = featuresTarget[matchIndex].descriptor;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches, float epsilon, float* fundamental, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;    
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold){
      match.invalid = true;
    }
    else{
      match.invalid = false;
      match.descriptors[0] = feature.descriptor;
      match.descriptors[1] = featuresTarget[matchIndex].descriptor;
      match.keyPoints[0].loc = feature.loc;
      match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
      match.keyPoints[0].parentId = queryImageID;
      match.keyPoints[1].parentId = targetImageID;
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches,
float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;    
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(match.distance/nearestSeed > relativeThreshold*relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.descriptors[0] = feature.descriptor;
        match.descriptors[1] = featuresTarget[matchIndex].descriptor;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches, float epsilon, float* fundamental,
float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;    
    match.distance = currentDist;
    if(match.distance >= absoluteThreshold || matchIndex == -1){
      match.invalid = true;
    }
    else{
      if(match.distance/nearestSeed > relativeThreshold*relativeThreshold){
        match.invalid = true;
      }
      else{
        match.invalid = false;
        match.descriptors[0] = feature.descriptor;
        match.descriptors[1] = featuresTarget[matchIndex].descriptor;
        match.keyPoints[0].loc = feature.loc;
        match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
        match.keyPoints[0].parentId = queryImageID;
        match.keyPoints[1].parentId = targetImageID;
      }
    }
    matches[blockId] = match;
  }
}


template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, uint2_pair* matches, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    uint2_pair match;
    if(currentDist >= absoluteThreshold){
      match = {{queryImageID,blockId},{queryImageID,blockId}};
    }
    else{
      match = {{queryImageID,blockId},{targetImageID,(unsigned int)matchIndex}};
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, uint2_pair* matches, float epsilon, float* fundamental, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    uint2_pair match;
    if(currentDist >= absoluteThreshold){
      match = {{queryImageID,blockId},{queryImageID,blockId}};
    }
    else{
      match = {{queryImageID,blockId},{targetImageID,(unsigned int)matchIndex}};
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, uint2_pair* matches, float* seedDistances, float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){
      currentDist = feature.descriptor.distProtocol(featuresTarget[f].descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    uint2_pair match;
    if(currentDist >= absoluteThreshold || matchIndex == -1){
      match = {{queryImageID,blockId},{queryImageID,blockId}};
    }
    else{
      if(currentDist/nearestSeed > relativeThreshold){
        match = {{queryImageID,blockId},{queryImageID,blockId}};
      }
      else{
      match = {{queryImageID,blockId},{targetImageID,(unsigned int)matchIndex}};
      }
    }
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, uint2_pair* matches, float epsilon, float* fundamental, float* seedDistances, 
float relativeThreshold, float absoluteThreshold){
  unsigned int blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[32];
    __shared__ float localDist[32];
    localMatch[threadIdx.x] = -1;
    float nearestSeed = seedDistances[blockId];
    localDist[threadIdx.x] = absoluteThreshold;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesTarget;
    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0]*feature.loc.x) + (fundamental[1]*feature.loc.y) + fundamental[2];
    epipolar.y = (fundamental[3]*feature.loc.x) + (fundamental[4]*feature.loc.y) + fundamental[5];
    epipolar.z = (fundamental[6]*feature.loc.x) + (fundamental[7]*feature.loc.y) + fundamental[8];

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 32){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) > regEpsilon) continue;
      currentDist = feature.descriptor.distProtocol(currentFeature.descriptor,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = absoluteThreshold;
    int matchIndex = -1;
    for(int i = 0; i < 32; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    uint2_pair match;
    if(currentDist >= absoluteThreshold || matchIndex == -1){
      match = {{queryImageID,blockId},{queryImageID,blockId}};
    }
    else{
      if(currentDist/nearestSeed > relativeThreshold){
        match = {{queryImageID,blockId},{queryImageID,blockId}};
      }
      else{
        match = {{queryImageID,blockId},{targetImageID,(unsigned int)matchIndex}};
      }
    }
    matches[blockId] = match;
  }
}


/*
  utility kernels
*/
__global__ void ssrlcv::convertMatchToRaw(unsigned long numMatches, ssrlcv::Match* rawMatches, ssrlcv::DMatch* matches){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if(globalID < numMatches){
    rawMatches[globalID] = Match(matches[globalID]);
  }
}
template<typename T>
__global__ void ssrlcv::convertMatchToRaw(unsigned long numMatches, ssrlcv::Match* rawMatches, ssrlcv::FeatureMatch<T>* matches){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if(globalID < numMatches){
    rawMatches[globalID] = Match(matches[globalID]);
  }
}
